
#include <hip/hip_runtime.h>
#include<stdio.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/rotationFeatureCuda.cu"
#else

void rotation_feature(THCTensor *feature,
                      THCTensor *output){

    THCUNN_assertSameGPU(state, 2, feature, output);

    const uint16 batch = feature->size[0];
    const uint16 channel = feature->size[1];
    const uint16 H = feature->size[2];
    const uint16 W = feature->size[3];

    const uint32 count = batch*channel*H*W;

    THCTensor_(resize4d)(state, output, batch, channel, H, W);
    real *featureData = THCTensor_(data)(state, feature);
    real *outputData = THCTensor_(data)(state, output);

    rotationFeature(THCState_getCurrentStream(state),
                    featureData,
                    outputData,
                    batch,
                    channel,
                    H,
                    count);
    THCudaCheck(cudaGetLastError());
}

void rotation_grad(THCTensor *grad,
                   THCTensor *output){
    THCUNN_assertSameGPU(state, 2, grad, output);

    const uint16 batch = grad->size[0];
    const uint16 channel = grad->size[1];
    const uint16 H = grad->size[2];
    const uint16 W = grad->size[3];
    const uint32 count = batch*channel*H*W;

    THCTensor_(resize4d)(state, output, batch, channel, H, W);
    real *gradData = THCTensor_(data)(state, grad);
    real *outputData = THCTensor_(data)(state, output);

    rotationGrad(THCState_getCurrentStream(state),
                    gradData,
                    outputData,
                    batch,
                    channel,
                    H,
                    count);
    THCudaCheck(cudaGetLastError());
}

#endif