#include "hip/hip_runtime.h"
#include"rotationFeatureKernel.h"
#include<stdio.h>

__global__ void rotationFeatureKernel(double* featureData,
                                        double* outputData,
                                        const uint16 batch,
                                        const uint16 channel,
                                        const uint16 H){
    const uint32 threadID = blockIdx.x * CUDA_NUM_THREADS + threadIdx.x;
    const uint16 channelNum = threadID / (H*H);
    const uint16 channelNo = channelNum % channel;
    const uint16 elementNum = threadID % (H*H);
    const uint16 rowNum = elementNum / H;
    const uint16 colNum = elementNum % H;

    if(channelNo == 1)
        *(outputData + channelNum*H*H + H*colNum+H-rowNum-1) = *(featureData+threadID);
    else if(channelNo == 2)
        *(outputData + channelNum*H*H + (H-rowNum-1)*H + H-colNum-1) = *(featureData+threadID);
    else if(channelNo == 3)
        *(outputData + threadID) = *(featureData + channelNum*H*H +  H*colNum+H-rowNum-1);
    else
        *(outputData+threadID) = *(featureData+threadID);
}

__global__ void rotationGradKernel(double* gradData,
                                   double* outputData,
                                   const uint16 batch,
                                   const uint16 channel,
                                   const uint16 H){
    const uint32 threadID = blockIdx.x * CUDA_NUM_THREADS + threadIdx.x;
    const uint16 channelNum = threadID / (H*H);
    const uint16 channelNo = channelNum % channel;
    const uint16 elementNum = threadID % (H*H);
    const uint16 rowNum = elementNum / H;
    const uint16 colNum = elementNum % H;

    if(channelNo == 1)
        *(outputData + threadID) = *(gradData + channelNum*H*H +  H*colNum+H-rowNum-1);
    else if(channelNo == 2)
        *(outputData + channelNum*H*H + (H-rowNum-1)*H + H-colNum-1) = *(gradData+threadID);
    else if(channelNo == 3)
        *(outputData + channelNum*H*H + H*colNum+H-rowNum-1) = *(gradData+threadID);
    else
        *(outputData+threadID) = *(gradData+threadID);
}

#ifdef __cplusplus
extern "C" {
#endif

void rotationFeature(hipStream_t stream,
                     double* featureData,
                     double* outputData,
                     const uint16 batch,
                     const uint16 channel,
                     const uint16 H,
                     const uint32 count){
    rotationFeatureKernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0>>>(featureData, outputData, batch, channel, H);
}

void rotationGrad(hipStream_t stream,
                    double* gradData,
                    double* outputData,
                    const uint16 batch,
                    const uint16 channel,
                    const uint16 H,
                    const uint32 count){
    rotationGradKernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0>>>(gradData, outputData, batch, channel, H);
}

#ifdef __cplusplus
}
#endif